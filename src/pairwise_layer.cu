#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>

#include "pairwise_layer.hpp"
#include "math_functions.hpp"

namespace caffe {

  template <typename Dtype>
  __global__ void ProdForward(const int M_, const int N_, const int K_,
                               const Dtype* bottom_data_a, const Dtype* bottom_data_b,
                               Dtype* top_data) {
    CUDA_KERNEL_LOOP(index, M_ * N_ * K_) {
      int m = index / N_ / K_;
      int nk = index % (N_* K_);
      int n = nk / K_;
      int k = nk % K_;
      top_data[index] = bottom_data_a[m * K_ + k] * bottom_data_b[n * K_ + k];
    }
  }

  template <typename Dtype>
  __global__ void SumForward(const int M_, const int N_, const int K_,
                              const Dtype* bottom_data_a, const Dtype* bottom_data_b,
                              Dtype coeff0, Dtype coeff1,
                              Dtype* top_data) {
    CUDA_KERNEL_LOOP(index, M_ * N_ * K_) {
      int m = index / N_ / K_;
      int nk = index % (N_* K_);
      int n = nk / K_;
      int k = nk % K_;
      top_data[index] = bottom_data_a[m * K_ + k] *coeff0 + bottom_data_b[n * K_ + k] * coeff1;
    }
  }

  template <typename Dtype>
  __global__ void MaxForward(const int M_, const int N_, const int K_,
                             const Dtype* bottom_data_a, const Dtype* bottom_data_b,
                             Dtype* top_data, int* mask) {
    CUDA_KERNEL_LOOP(index, M_ * N_ * K_) {
      int m = index / N_ / K_;
      int nk = index % (N_* K_);
      int n = nk / K_;
      int k = nk % K_;
      if (bottom_data_a[m * K_ + k] > bottom_data_b[n * K_ + k]) {
        top_data[index] = bottom_data_a[m * K_ + k];
        mask[index] = 0;
      }
      else {
        top_data[index] = bottom_data_b[n * K_ + k];
        mask[index] = 1;
      }
    }
  }

template <typename Dtype>
void PairwiseLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int* mask = NULL;
  const int count = top[0]->count();
  Dtype* top_data = top[0]->mutable_gpu_data();
  switch (op_) {
  case PairwiseParameter_PairwiseOp_PROD:
    // NOLINT_NEXT_LINE(whitespace/operators)
    ProdForward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
      M_, N_, K_, bottom[0]->gpu_data(), bottom[1]->gpu_data(), top_data);
    break;
  case PairwiseParameter_PairwiseOp_SUM:
    // NOLINT_NEXT_LINE(whitespace/operators)
    SumForward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
      M_, N_, K_, bottom[0]->gpu_data(), bottom[1]->gpu_data(), coeffs_[0], coeffs_[1], top_data);
    break;
  case PairwiseParameter_PairwiseOp_MAX:
    mask = max_idx_.mutable_gpu_data();
    // NOLINT_NEXT_LINE(whitespace/operators)
    MaxForward<Dtype> <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      M_, N_, K_, bottom[0]->gpu_data(), bottom[1]->gpu_data(), top_data, mask);
    break;
  default:
    LOG(FATAL) << "Unknown elementwise operation.";
  }
}



INSTANTIATE_LAYER_GPU_FUNCS(PairwiseLayer);

}  // namespace caffe
