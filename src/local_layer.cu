#include "hip/hip_runtime.h"
#include <vector>

#include "filler.hpp"
#include "layer.hpp"
#include "im2col.hpp"
#include "math_functions.hpp"
#include "custom_layers.hpp"

namespace caffe {



template <typename Dtype>
__global__ void local_update1_gpu_kernel(const Dtype* data_A, const Dtype* data_B,
                                    Dtype* data_R, const int filter_num,
                                    const int location_num, const int output_num) {
  int total = filter_num * location_num * output_num;
  CUDA_KERNEL_LOOP(index, total) {
    int p = index % location_num;
    int n = (index / location_num) % filter_num;
    int q = (index / location_num) / filter_num;
    data_R[index] += data_A[q*location_num+p] * data_B[n*location_num+p];
  }
}

template <typename Dtype>
void local_update1_gpu(const Dtype* data_A, const Dtype* data_B,
                       Dtype* data_R, const int filter_num,
                       const int location_num, const int output_num) {
  // data_A is output_num x location_num
  // data_B is filter_num x location_num
  // data_R is output_num x filter_num x location_num, the update performed is Rqnp += Aqp * Bnp

  // NOLINT_NEXT_LINE(whitespace/operators)
  local_update1_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(filter_num * location_num * output_num),
                             CAFFE_CUDA_NUM_THREADS>>>(data_A, data_B, data_R, filter_num, location_num, output_num);
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void local_update1_gpu<float>(const float* data_A, const float* data_B,
                                float* data_R, const int filter_num,
                                const int location_num, const int output_num);
template void local_update1_gpu<double>(const double* data_A, const double* data_B,
                                double* data_R, const int filter_num,
                                const int location_num, const int output_num);


template <typename Dtype>
__global__ void local_update2_gpu_kernel(const Dtype* data_A, const Dtype* data_B,
                                Dtype* data_R, const int filter_num,
                                const int location_num, const int output_num) {
  int total = filter_num * location_num;
  CUDA_KERNEL_LOOP(index, total) {
    int p = index % location_num;
    int n = (index / location_num);
    for (int q=0; q<output_num; q++) {
      data_R[index] += data_A[q*location_num+p] * data_B[(q*filter_num+n)*location_num+p];
    }
  }
}

template <typename Dtype>
void local_update2_gpu(const Dtype* data_A, const Dtype* data_B,
                       Dtype* data_R, const int filter_num,
                       const int location_num, const int output_num) {
  // data_A is output_num x location_num
  // data_B is output_num x filter_num x location_num
  // data_R is filter_num x location_num, the update performed is Rnp += \sum_q(Aqp * Bqnp)

  // NOLINT_NEXT_LINE(whitespace/operators)
  local_update2_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(filter_num * location_num),
                             CAFFE_CUDA_NUM_THREADS>>>(data_A, data_B, data_R, filter_num, location_num, output_num);
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void local_update2_gpu<float>(const float* data_A, const float* data_B,
                       float* data_R, const int filter_num,
                       const int location_num, const int output_num);
template void local_update2_gpu<double>(const double* data_A, const double* data_B,
                       double* data_R, const int filter_num,
                       const int location_num, const int output_num);




/// @brief refer to CPU forward -- the BLAS implementation is the same.
template <typename Dtype>
void LocalLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {

  Dtype* x_data = col_buffer_.mutable_gpu_data();
  const Dtype* weight = this->blobs_[0]->gpu_data();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();

  Blob<Dtype> E;
  E.Reshape(1, 1, 1, K_);
  FillerParameter filler_param;
  filler_param.set_value(1);
  ConstantFiller<Dtype> filler(filler_param);
  filler.Fill(&E);

  Blob<Dtype> intermediate;
  intermediate.Reshape(1, 1, K_, N_);
  for (int n=0; n<num_; n++) {
    im2col_gpu(bottom_data + bottom[0]->offset(n), channels_, height_,
               width_, kernel_size_, kernel_size_, pad_, pad_, stride_, stride_, 1, 1, x_data);

    for (int m=0; m<num_output_; m++) {
      caffe_gpu_mul(K_*N_, x_data, weight+this->blobs_[0]->offset(m),
                    intermediate.mutable_gpu_data());

      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, 1, N_, K_,
                            (Dtype)1., E.gpu_data(), intermediate.gpu_data(),
                            (Dtype)0., top_data + top[0]->offset(n, m));
    }

    if (bias_term_) {
      caffe_gpu_add(M_ * N_, this->blobs_[1]->gpu_data(),
                    top_data + top[0]->offset(n),
                    top_data + top[0]->offset(n));
    }
  }

}

INSTANTIATE_LAYER_GPU_FUNCS(LocalLayer);

}  // namespace caffe