#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "neuron_layer.hpp"
#include "cosine_layer.hpp"

namespace caffe {

  // CUDA kernele for forward
  template <typename Dtype>
  __global__ void CosineForward(const int n, const Dtype* in, Dtype* out) {
    CUDA_KERNEL_LOOP(index, n) {
      out[index] = cos(in[index]);
    }
  }

  // CUDA kernel for bottom backward
  template <typename Dtype>
  __global__ void CosineBackward(const int n, const Dtype* in_diff, const Dtype* in_data, Dtype* out_diff) {
    CUDA_KERNEL_LOOP(index, n) {
      out_diff[index] = in_diff[index] * -1 * sin(in_data[index]);
    }
  }

  template <typename Dtype>
  void CosineLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                      const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    const int count = bottom[0]->count();

    // NOLINT_NEXT_LINE(whitespace/operators)
    CosineForward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
      count, bottom_data, top_data);
    CUDA_POST_KERNEL_CHECK;
  }

  INSTANTIATE_LAYER_GPU_FUNCS(CosineLayer);


}  // namespace caffe
