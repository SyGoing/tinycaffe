#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "neuron_layer.hpp"
#include "cosine_layer.hpp"

namespace caffe {

  // CUDA kernele for forward
  template <typename Dtype>
  __global__ void CosineForward(const int n, const Dtype* in, Dtype* out) {
    CUDA_KERNEL_LOOP(index, n) {
      out[index] = cos(in[index]);
    }
  }

  // CUDA kernel for bottom backward
  template <typename Dtype>
  __global__ void CosineBackward(const int n, const Dtype* in_diff, const Dtype* in_data, Dtype* out_diff) {
    CUDA_KERNEL_LOOP(index, n) {
      out_diff[index] = in_diff[index] * -1 * sin(in_data[index]);
    }
  }

  template <typename Dtype>
  void CosineLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                      const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    const int count = bottom[0]->count();

    // NOLINT_NEXT_LINE(whitespace/operators)
    CosineForward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
      count, bottom_data, top_data);
    CUDA_POST_KERNEL_CHECK;
  }

  template <typename Dtype>
  void CosineLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                       const vector<bool>& propagate_down,
                                       const vector<Blob<Dtype>*>& bottom) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    const int count = bottom[0]->count();

    // Propagate to bottom
    if (propagate_down[0]) {
      Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
      // NOLINT_NEXT_LINE(whitespace/operators)
      CosineBackward<Dtype> << <CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS >> >(
          count, top_diff, bottom_data, bottom_diff);
      CUDA_POST_KERNEL_CHECK;
    }
  }


  INSTANTIATE_LAYER_GPU_FUNCS(CosineLayer);


}  // namespace caffe
