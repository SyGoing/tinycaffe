
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "../layer.hpp"
#include "../math_functions.hpp"
#include "../filler.hpp"
#include "cudnn_bn_layer.hpp"

#if CUDNN_VERSION_MIN(4, 0, 0)

namespace caffe {

  template <typename Dtype>
  void CuDNNBNLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                        const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    const Dtype* scale_data = this->blobs_[0]->gpu_data();
    const Dtype* bias_data = this->blobs_[1]->gpu_data();

    if (this->phase_ == TEST) {
      const Dtype* running_mean_data = this->blobs_[2]->gpu_data();
      const Dtype* running_inv_variance_data = this->blobs_[3]->gpu_data();
      CUDNN_CHECK(cudnnBatchNormalizationForwardInference(handle_,
                                                          CUDNN_BATCHNORM_SPATIAL,
                                                          cudnn::dataType<Dtype>::one,
                                                          cudnn::dataType<Dtype>::zero,
                                                          bottom_desc_, bottom_data,
                                                          top_desc_, top_data,
                                                          bn_param_desc_, scale_data, bias_data,
                                                          running_mean_data, running_inv_variance_data,
                                                          this->bn_eps_));
    }
    else {
      Dtype* running_mean_data = this->blobs_[2]->mutable_gpu_data();
      Dtype* running_inv_variance_data = this->blobs_[3]->mutable_gpu_data();
      Dtype* save_mean_data = save_mean_.mutable_gpu_data();
      Dtype* save_inv_variance_data = save_inv_variance_.mutable_gpu_data();
      CUDNN_CHECK(cudnnBatchNormalizationForwardTraining(handle_,
                                                         CUDNN_BATCHNORM_SPATIAL,
                                                         cudnn::dataType<Dtype>::one,
                                                         cudnn::dataType<Dtype>::zero,
                                                         bottom_desc_, bottom_data,
                                                         top_desc_, top_data,
                                                         bn_param_desc_, scale_data, bias_data,
                                                         this->bn_momentum_,
                                                         running_mean_data, running_inv_variance_data,
                                                         this->bn_eps_,
                                                         save_mean_data, save_inv_variance_data));
    }
  }


  INSTANTIATE_LAYER_GPU_FUNCS(CuDNNBNLayer);

}  // namespace caffe
#endif
#endif
