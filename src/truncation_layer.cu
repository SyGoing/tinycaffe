#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "truncation_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void TruncationForward(const int n, const Dtype* in, Dtype* out,
    Dtype lower_bound, Dtype upper_bound) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = min(max(in[index], lower_bound), upper_bound);
  }
}

template <typename Dtype>
void TruncationLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  Dtype lower_bound = this->layer_param_.truncation_param().lower_bound();
  Dtype upper_bound = this->layer_param_.truncation_param().upper_bound();
  // NOLINT_NEXT_LINE(whitespace/operators)
  TruncationForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, lower_bound, upper_bound);
  CUDA_POST_KERNEL_CHECK;
}



INSTANTIATE_LAYER_GPU_FUNCS(TruncationLayer);


}  // namespace caffe
