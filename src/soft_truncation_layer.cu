#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "soft_truncation_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SoftTruncationForward(const int n, const Dtype* in, Dtype* out,
    Dtype c) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = 1 - exp(in[index] / (-c));
  }
}

template <typename Dtype>
void SoftTruncationLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  Dtype c = this->layer_param_.soft_truncation_param().c();
  // NOLINT_NEXT_LINE(whitespace/operators)
  SoftTruncationForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, c);
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(SoftTruncationLayer);


}  // namespace caffe
