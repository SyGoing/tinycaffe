#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "soft_truncation_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SoftTruncationForward(const int n, const Dtype* in, Dtype* out,
    Dtype c) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = 1 - exp(in[index] / (-c));
  }
}

template <typename Dtype>
void SoftTruncationLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  Dtype c = this->layer_param_.soft_truncation_param().c();
  // NOLINT_NEXT_LINE(whitespace/operators)
  SoftTruncationForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, c);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void SoftTruncationBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff, Dtype c) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * (1 - in_data[index]) / c;
  }
}

template <typename Dtype>
void SoftTruncationLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    Dtype c = this->layer_param_.soft_truncation_param().c();
    // NOLINT_NEXT_LINE(whitespace/operators)
    SoftTruncationBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, top_data, bottom_diff, c);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(SoftTruncationLayer);


}  // namespace caffe
