#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "layer.hpp"
#include "math_functions.hpp"
#include "channel_scale_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void kernel_channel_scale(const int num, const int channels, const int spatial_dim,
                                     Dtype alpha, const Dtype* data, const Dtype* norm_data,
                                     Dtype beta, Dtype* output_data) {
  CUDA_KERNEL_LOOP(index, num * channels * spatial_dim) {
    int n = index / channels / spatial_dim;
    int s = index % spatial_dim;
    output_data[index] = alpha * data[index] * norm_data[n * spatial_dim + s] + beta * output_data[index];
  }
}

template <typename Dtype>
__global__ void kernel_channel_sum(const int num, const int channels, const int spatial_dim,
                                   const Dtype* data, Dtype* sum_data) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int n = index / spatial_dim;
    int s = index % spatial_dim;
    Dtype sum = 0;
    for (int c = 0; c < channels; ++c) {
      sum += data[(n * channels + c) * spatial_dim + s];
    }
    sum_data[index] = sum;
  }
}

template <typename Dtype>
void ChannelScaleLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* scale_data = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  
  if (do_forward_) {
    if (global_scale_) {
      int count = bottom[0]->count();
      Dtype* scale = this->blobs_[0]->mutable_cpu_data();
      Dtype mean_norm = bottom[1]->asum_data() / (Dtype)bottom[1]->count();
      if (this->phase_ == TRAIN) {
        if (scale[0] < 0) {
          scale[0] = mean_norm;
        }
        else {
          scale[0] = scale[0] * 0.99 + mean_norm * 0.01;
        }
        scale[0] = std::min(scale[0], max_global_scale_);
        scale[0] = std::max(scale[0], min_global_scale_);
      }
      if (top.size() == 2) {
        top[1]->mutable_cpu_data()[0] = scale[0];
      }
      caffe_gpu_scale(count, scale[0], bottom_data, top_data);
    }
    else {
      int num = bottom[0]->num();
      int channels = bottom[0]->channels();
      int spatial_dim = bottom[0]->height() * bottom[0]->width();
      // NOLINT_NEXT_LINE(whitespace/operators)
      kernel_channel_scale<Dtype> << <CAFFE_GET_BLOCKS(num*channels*spatial_dim),
        CAFFE_CUDA_NUM_THREADS >> >(num, channels, spatial_dim, Dtype(1), bottom_data, scale_data, Dtype(0), top_data);
    }
  }
  else {
    caffe_copy(bottom[0]->count(), bottom_data, top_data);
  }
}




INSTANTIATE_LAYER_GPU_FUNCS(ChannelScaleLayer);


}  // namespace caffe