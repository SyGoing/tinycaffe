#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "layer.hpp"
#include "math_functions.hpp"
#include "channel_scale_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void kernel_channel_scale(const int num, const int channels, const int spatial_dim,
                                     Dtype alpha, const Dtype* data, const Dtype* norm_data,
                                     Dtype beta, Dtype* output_data) {
  CUDA_KERNEL_LOOP(index, num * channels * spatial_dim) {
    int n = index / channels / spatial_dim;
    int s = index % spatial_dim;
    output_data[index] = alpha * data[index] * norm_data[n * spatial_dim + s] + beta * output_data[index];
  }
}

template <typename Dtype>
__global__ void kernel_channel_sum(const int num, const int channels, const int spatial_dim,
                                   const Dtype* data, Dtype* sum_data) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int n = index / spatial_dim;
    int s = index % spatial_dim;
    Dtype sum = 0;
    for (int c = 0; c < channels; ++c) {
      sum += data[(n * channels + c) * spatial_dim + s];
    }
    sum_data[index] = sum;
  }
}

template <typename Dtype>
void ChannelScaleLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* scale_data = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  
  if (do_forward_) {
    if (global_scale_) {
      int count = bottom[0]->count();
      Dtype* scale = this->blobs_[0]->mutable_cpu_data();
      Dtype mean_norm = bottom[1]->asum_data() / (Dtype)bottom[1]->count();
      if (this->phase_ == TRAIN) {
        if (scale[0] < 0) {
          scale[0] = mean_norm;
        }
        else {
          scale[0] = scale[0] * 0.99 + mean_norm * 0.01;
        }
        scale[0] = std::min(scale[0], max_global_scale_);
        scale[0] = std::max(scale[0], min_global_scale_);
      }
      if (top.size() == 2) {
        top[1]->mutable_cpu_data()[0] = scale[0];
      }
      caffe_gpu_scale(count, scale[0], bottom_data, top_data);
    }
    else {
      int num = bottom[0]->num();
      int channels = bottom[0]->channels();
      int spatial_dim = bottom[0]->height() * bottom[0]->width();
      // NOLINT_NEXT_LINE(whitespace/operators)
      kernel_channel_scale<Dtype> << <CAFFE_GET_BLOCKS(num*channels*spatial_dim),
        CAFFE_CUDA_NUM_THREADS >> >(num, channels, spatial_dim, Dtype(1), bottom_data, scale_data, Dtype(0), top_data);
    }
  }
  else {
    caffe_copy(bottom[0]->count(), bottom_data, top_data);
  }
}



template <typename Dtype>
void ChannelScaleLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* scale_data = bottom[1]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  Dtype* scale_diff = bottom[1]->mutable_gpu_diff();

  int num = top[0]->num();
  int channels = top[0]->channels();
  int spatial_dim = bottom[0]->height() * bottom[0]->width();

  if (propagate_down[1]) {
    if (do_backward_scale_) {
      caffe_gpu_mul(bottom[0]->count(), top_diff, bottom_data, bottom_diff);
      // NOLINT_NEXT_LINE(whitespace/operators)
      kernel_channel_sum<Dtype> << <CAFFE_GET_BLOCKS(num*spatial_dim),
        CAFFE_CUDA_NUM_THREADS >> >(num, channels, spatial_dim, bottom_diff, scale_diff);
    }
    else {
      caffe_gpu_set(bottom[1]->count(), Dtype(0), scale_diff);
    }
  }
  
  if (propagate_down[0]) {
    if (do_backward_feature_) {
      // NOLINT_NEXT_LINE(whitespace/operators)
      kernel_channel_scale<Dtype> << <CAFFE_GET_BLOCKS(num*channels*spatial_dim),
        CAFFE_CUDA_NUM_THREADS >> >(num, channels, spatial_dim, Dtype(1), top_diff, scale_data, Dtype(0), bottom_diff);
    }
    else {
      caffe_copy(bottom[0]->count(), top_diff, bottom_diff);
    }
  }

  //if (global_scale_ && this->param_propagate_down_[0]) {
  //  caffe_gpu_dot(bottom[0]->count(), top_diff, bottom_data, this->blobs_[0]->mutable_cpu_diff());
  //}
}

INSTANTIATE_LAYER_GPU_FUNCS(ChannelScaleLayer);


}  // namespace caffe