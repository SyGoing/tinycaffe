#include "hip/hip_runtime.h"
#include <vector>

#include "filler.hpp"
#include "inner_product_layer.hpp"
#include "math_functions.hpp"

namespace caffe {

  template <typename Dtype>
  __global__ void kernel_channel_dot(const int num, const int dim,
                                     const Dtype* data_1, const Dtype* data_2,
                                     Dtype* channel_dot, Dtype epsilon) {
    CUDA_KERNEL_LOOP(index, num) {
      Dtype dot = 0;
      for (int d = 0; d < dim; ++d) {
        dot += data_1[index * dim + d] * data_2[index * dim + d];
      }
      channel_dot[index] = dot + epsilon;
    }
  }

  template <typename Dtype>
  __global__ void kernel_channel_scal(const int num, const int dim,
                                      const Dtype* norm_data,
                                      Dtype* input_output_data) {
    CUDA_KERNEL_LOOP(index, num * dim) {
      int n = index / dim;
      input_output_data[index] *= norm_data[n];
    }
  }

template <typename Dtype>
void InnerProductLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const Dtype* weight = bottom.size() >= 2 ? bottom[1]->gpu_data() : this->blobs_[0]->gpu_data();

  if (normalize_ && bottom.size() == 1) {
    Dtype* mutable_weight = this->blobs_[0]->mutable_gpu_data();
    Dtype* weight_norm_data = weight_norm_.mutable_gpu_data();
    // NOLINT_NEXT_LINE(whitespace/operators)
    kernel_channel_dot<Dtype> << <CAFFE_GET_BLOCKS(N_),
      CAFFE_CUDA_NUM_THREADS >> >(N_, K_, weight, weight, weight_norm_data, 1e-12);
    caffe_gpu_powx(N_, weight_norm_data, Dtype(-0.5), weight_norm_data);
    // NOLINT_NEXT_LINE(whitespace/operators)
    kernel_channel_scal<Dtype> << <CAFFE_GET_BLOCKS(N_ * K_),
      CAFFE_CUDA_NUM_THREADS >> >(N_, K_, weight_norm_data, mutable_weight);
  }

  if (M_ == 1) {
    caffe_gpu_gemv<Dtype>(CblasNoTrans, N_, K_, (Dtype)1.,
                         weight, bottom_data, (Dtype)0., top_data);
    if (bias_term_)
      caffe_gpu_axpy<Dtype>(N_, bias_multiplier_.cpu_data()[0],
                            bottom.size() == 3 ? bottom[2]->gpu_data() : this->blobs_[1]->gpu_data(), top_data);
  } else {
    caffe_gpu_gemm<Dtype>(CblasNoTrans,
                          transpose_ ? CblasNoTrans : CblasTrans,
                          M_, N_, K_, (Dtype)1.,
                          bottom_data, weight, (Dtype)0., top_data);
    if (bias_term_)
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, (Dtype)1.,
                            bias_multiplier_.gpu_data(),
                            bottom.size() == 3 ? bottom[2]->gpu_data() : this->blobs_[1]->gpu_data(), (Dtype)1., top_data);
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(InnerProductLayer);

}  // namespace caffe
