#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "layer.hpp"
#include "custom_layers.hpp"

namespace caffe {

// CUDA kernele for forward
template <typename Dtype>
__global__ void InsanityForwardTrain(const int n, const int channels, const int dim,
    const Dtype* in, Dtype* out, const Dtype* slope_data) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0 ? in[index] : in[index] / slope_data[index];
  }
}

// CUDA kernele for forward
template <typename Dtype>
__global__ void InsanityForwardTest(const int n, const int channels, const int dim,
    const Dtype* in, Dtype* out, const Dtype slope_data) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0 ? in[index] : in[index] / slope_data;
  }
}

// CUDA kernel for bottom backward
template <typename Dtype>
__global__ void InsanityBackward(const int n, const int channels, const int dim,
    const Dtype* in_diff, const Dtype* in_data, Dtype* out_diff,
    const Dtype* slope_data) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * ((in_data[index] > 0)
        + (in_data[index] <= 0) / slope_data[index]);
  }
}

// CUDA kernel for bottom backward
template <typename Dtype>
__global__ void InsanityBackwardTest(const int n, const int channels, const int dim,
                                 const Dtype* in_diff, const Dtype* in_data, Dtype* out_diff,
                                 const Dtype slope_data) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * ((in_data[index] > 0)
                                        + (in_data[index] <= 0) / slope_data);
  }
}

template <typename Dtype>
void InsanityLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  const int dim = bottom[0]->count(2);
  const int channels = bottom[0]->channels();

  // For in-place computation
  if (top[0] == bottom[0] && lb_ < 0) {
    caffe_copy(count, bottom_data, bottom_memory_.mutable_gpu_data());
  }

  if (this->phase_ == TRAIN) {
    Dtype* slope_data =
        static_cast<Dtype*>(alpha.mutable_gpu_data());
	caffe_gpu_rng_uniform<Dtype>(count, lb_, ub_, slope_data);
    // set thresholds
    // NOLINT_NEXT_LINE(whitespace/operators)
	InsanityForwardTrain<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, channels, dim, bottom_data, top_data, slope_data);
    CUDA_POST_KERNEL_CHECK;
  } else {
    InsanityForwardTest<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
		count, channels, dim, bottom_data, top_data, mean_slope);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(InsanityLayer);


}  // namespace caffe
